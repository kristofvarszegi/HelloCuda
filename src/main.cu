
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 16; 
const int blocksize = 16; 

__global__
void hello(char *a, int *b) {
    printf("before: %c", a[threadIdx.x]);
    a[threadIdx.x] += b[threadIdx.x];
    printf("after: %c", a[threadIdx.x]);
}


int main()
{
    char a[N] = "Hello \0\0\0\0\0\0";
    int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    char *ad;
    int *bd;
    const int csize = N*sizeof(char);
    const int isize = N*sizeof(int);

    printf("%s\n", a);

    hipMalloc((void**)&ad, csize);
    hipMalloc((void**)&bd, isize);
    hipMemcpy( ad, a, csize, hipMemcpyHostToDevice); 
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice); 

    dim3 dimBlock(blocksize, 1);
    dim3 dimGrid(1, 1);
    hello<<<dimGrid, dimBlock>>>(ad, bd);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost); 
    hipFree(ad);
    hipFree(bd);

    printf("%s\n", a);
    return 0;

}